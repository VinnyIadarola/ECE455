
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloKernel() {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from thread %i\n", thread_id);
} 



int main() {
    const int num_threads = 4;
    helloKernel<<<1, num_threads>>>();
    hipDeviceSynchronize();
    return 0;

}