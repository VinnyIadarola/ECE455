#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 256

__global__ void reduce_shared(const int* in, int* out, size_t n_elems) {
    __shared__ int sdata[BLOCK_DIM];

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int x = (idx < n_elems) ? in[idx] : 0;
    sdata[tid] = x;
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride)
            sdata[tid] += sdata[tid + stride];
        __syncthreads();
    }
    if (tid == 0)
        atomicAdd(out, sdata[0]);
}

static void checkCuda(hipError_t err, const char* msg = "") {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error %s: %s\n", msg, hipGetErrorString(err));
        exit(1);
    }
}

int main() {
    const size_t N = 1 << 20; // 1M elements
    size_t bytes = N * sizeof(int);

    int *h_in = (int*)malloc(bytes);
    for (size_t i = 0; i < N; ++i) h_in[i] = 1; // expected sum = N

    int *d_in = nullptr, *d_out = nullptr;
    checkCuda(hipMalloc(&d_in, bytes));
    checkCuda(hipMalloc(&d_out, sizeof(int)));
    checkCuda(hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice));
    checkCuda(hipMemset(d_out, 0, sizeof(int)));

    dim3 block(BLOCK_DIM);
    dim3 grid((N + BLOCK_DIM - 1) / BLOCK_DIM);

    reduce_shared<<<grid, block>>>(d_in, d_out, N);
    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());

    int h_out = 0;
    checkCuda(hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost));

    printf("device sum = %d, expected = %zu\n", h_out, N);

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    return 0;
}