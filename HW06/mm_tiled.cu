#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

template <typename T>
__global__ void mm_tiled(const T* A, const T* B, T* C, int N) {
    __shared__ T tile_A[TILE_SIZE][TILE_SIZE];
    __shared__ T tile_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    T val = 0;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < N && (t * TILE_SIZE + threadIdx.x) < N)
            tile_A[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        else
            tile_A[threadIdx.y][threadIdx.x] = 0;

        if (col < N && (t * TILE_SIZE + threadIdx.y) < N)
            tile_B[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        else
            tile_B[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k)
            val += tile_A[threadIdx.y][k] * tile_B[k][threadIdx.x];

        __syncthreads();
    }

    if (row < N && col < N)
        C[row * N + col] = val;
}

int main() {
    const int N = 32;
    size_t bytes = N * N * sizeof(float);

    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    mm_tiled<float><<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    printf("C[0] = %f\n", h_C[0]);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}